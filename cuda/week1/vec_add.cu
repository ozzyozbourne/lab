#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(err, msg) \
    if (err != hipSuccess) { \
        fprintf(stderr, "%s (error code %s)!\n", msg, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    }

__global__ void vectorAdd(float *const A, float *const B, float *const C, const int num_elements) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < num_elements) { C[i] = A[i] + B[i]; }
}

int main(void) {
    hipError_t err = hipSuccess;

    const int num_elements = 5000, threads_per_block = 256, block_per_grid = (num_elements + threads_per_block - 1) / threads_per_block;
    const size_t size = num_elements * sizeof(float), total_size = 3 * size;

    float *const h_data = (float*) malloc(total_size);
    if (h_data == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    float *const h_A = h_data, *const h_B = h_data + num_elements, *const h_C = h_data + num_elements * 2;

    //Initing the data
    for(int i = 0; i < num_elements; i++) {
        h_A[i] = rand() / (float) RAND_MAX;
        h_B[i] = rand() / (float) RAND_MAX;
    }

    // Device allcation 
    float *d_data = NULL;
    err = hipMalloc((void **)&d_data, total_size);
    CUDA_CHECK(err, "Failed to allocate device vector");

    float *const d_A = d_data, *const d_B = d_data + num_elements, *const d_C = d_data + num_elements * 2;

    err = hipMemcpy(d_A, h_A, 2 * size, hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Failed to copy vector form host to device");

    vectorAdd<<<block_per_grid, threads_per_block>>>(d_A, d_B, d_C, num_elements);
    CUDA_CHECK(hipGetLastError(),"Failed to lauch vectorAdd kernel");

    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "Failed to copy vector c from device to host");

    for(int i = 0; i < num_elements; i++) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");
    hipFree(d_data);
    free(h_data);

    return 0;
}
